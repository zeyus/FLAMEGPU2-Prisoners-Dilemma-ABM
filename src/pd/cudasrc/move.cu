#include "hip/hip_runtime.h"
FLAMEGPU_AGENT_FUNCTION(move, flamegpu::MessageSpatial2D, flamegpu::MessageNone) {
    const flamegpu::id_t ID = FLAMEGPU->getID();
    const float REPULSE_FACTOR = FLAMEGPU->environment.getProperty<float>("repulse");
    const float RADIUS = FLAMEGPU->message_in.radius();
    float fx = 0.0;
    float fy = 0.0;
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    int count = 0;
    for (const auto &message : FLAMEGPU->message_in(x1, y1)) {
        if (message.getVariable<flamegpu::id_t>("id") != ID) {
            const float x2 = message.getVariable<float>("x");
            const float y2 = message.getVariable<float>("y");
            float x21 = x2 - x1;
            float y21 = y2 - y1;
            const float separation = sqrt(x21*x21 + y21*y21);
            if (separation < RADIUS && separation > 0.0f) {
                float k = sinf((separation / RADIUS)*3.141*-2)*REPULSE_FACTOR;
                // Normalise without recalculating separation
                x21 /= separation;
                y21 /= separation;
                fx += k * x21;
                fy += k * y21;
                count++;
            }
        }
    }
    fx /= count > 0 ? count : 1;
    fy /= count > 0 ? count : 1;
    FLAMEGPU->setVariable<float>("x", x1 + fx);
    FLAMEGPU->setVariable<float>("y", y1 + fy);
    FLAMEGPU->setVariable<float>("drift", sqrt(fx*fx + fy*fy));
    return flamegpu::ALIVE;
}